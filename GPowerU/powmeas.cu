#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <iomanip>
#include <math.h>
#include "nvml.h"
#include "hip/hip_runtime_api.h"
#include "GPowerU.hpp"
     
#include <stdio.h>
//#include <time.h>
//#include <sys/time.h>


//Output: data directory created, with a single text file per GPU (named hostname+GPU_uuid) with two semicolon-separated columns: Timestamp [us];Power measure [W]

// CUDA kernel to add elements of two arrays with loops designed to stress the GPU
// and extend the kernel execution
 __global__
 void add(int n, float *x, float *y)
  	{
      int index = blockIdx.x * blockDim.x + threadIdx.x;
      int stride = blockDim.x * gridDim.x;
    	take_GPU_time(); //Checkpoint power measure __device__ function
      __syncthreads();
      
      
      for (int i = index; i < n; i += stride){
      	for(int k=0; k<1000; k++) y[i] = x[i] + y[i];
        }	
       
      __syncthreads();
      
      take_GPU_time(); //Checkpoint power measure __device__ function
      for (int i = index; i < n; i += stride){
      	for(int k=0; k<1000; k++) y[i] = x[i] + y[i];
      }
      __syncthreads();
      
      take_GPU_time(); //Checkpoint power measure __device__ function
      for (int i = index; i < n; i += stride){
      	for(int k=0; k<10000; k++) y[i] = x[i] + y[i];
        }
      __syncthreads();
		
		
      take_GPU_time(true); //Checkpoint power measure __device__ function (last)
    } 
 


int blockSize = 256;
int numBlocks = (64 + blockSize - 1) / blockSize;

 
     
int main( int argc, char** argv)
    {	
    		 
//Initializations ==> enable the NVML library, starts CPU thread for the power monitoring,  
	if ( GPowerU_init() != 0 ) {
		fprintf ( stderr, "%s: error: initializing...\n", argv[0] );
			_exit (1);
	}
#if MULTIGPU_DISABLED	
      	int N =1<<20;
      	float *x, *y;
      
      	// Allocate Unified Memory -- accessible from CPU or GPU
      	checkCudaErrors(hipMallocManaged((void **)&x, N*sizeof(float)));
      	checkCudaErrors(hipMallocManaged((void **)&y, N*sizeof(float)));
     	   	
      	// Initialize x and y arrays on the Host
      	for (int i = 0; i < N; i++) {
        	x[i] = 1.0f;
        	y[i] = 2.0f;
      	}		
      
      	//Launch the GPU kernel
			
      	add<<<numBlocks, blockSize>>>(N, x, y);
      
       	//Checkpoint power measure CPU function ==> it calls its own hipDeviceSynchronize() 
     		GPowerU_checkpoints();
      	// Free memory
      	checkCudaErrors(hipFree(x));
      	checkCudaErrors(hipFree(y));
     	
#else
		//Add here your executable/script
		system("python ....py");
#endif     	
     	//Ends power monitoring, returns data output files
      	if ( GPowerU_end(2) != 0 )
	{
		fprintf ( stderr, " error: terminating...\n" );
		_exit (1);
	}
     	
      

      return 0;
    }
